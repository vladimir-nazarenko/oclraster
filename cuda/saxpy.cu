#include "hip/hip_runtime.h"
#include "saxpy.h"
#include <hip/hip_runtime.h>

#include <cuda/check_error.h>

__global__ void cuda_saxpy(float a, const float *x, const float *y, unsigned int sz, float *res)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < sz) {
        res[i] = a * x[i] + y[i];
    }
}

void saxpy(float a, const float *x, const float *y, unsigned int sz, float *res)
{
//    for (unsigned int i = 0; i < sz; ++i) {
//        res[i] = a * x[i] + y[i];
//    }

    float *d_x, *d_y, *d_res;

    cudaCheckError(hipMalloc(&d_x, sizeof(float) * sz));
    cudaCheckError(hipMalloc(&d_y, sizeof(float) * sz));
    cudaCheckError(hipMalloc(&d_res, sizeof(float) * sz));

    cudaCheckError(hipMemcpy(d_x, x, sizeof(float) * sz, hipMemcpyKind::hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(d_y, y, sizeof(float) * sz, hipMemcpyKind::hipMemcpyHostToDevice));

    int wgr_sz = 256;
    int grid_sz = (sz + wgr_sz - 1) / wgr_sz;

    cuda_saxpy<<<grid_sz, wgr_sz>>>(a, d_x, d_y, sz, d_res);

    cudaCheckError(hipMemcpy(res, d_res, sizeof(float) * sz, hipMemcpyKind::hipMemcpyDeviceToHost));

    cudaCheckError(hipFree(d_x));
    cudaCheckError(hipFree(d_y));
    cudaCheckError(hipFree(d_res));
}